
#include "hip/hip_runtime.h"
#include ""
#include <algorithm>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <Windows.h>

void MatrixRandomFill(unsigned long long *matrix, unsigned int m, unsigned int n)
{
	unsigned long long *dev_matrix = 0;
	hipSetDevice(0);
	hipMalloc((void**)&dev_matrix, m * n * sizeof(long long));

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64);
	hiprandSetGeneratorOffset(gen, GetTickCount64());
	hiprandSetQuasiRandomGeneratorDimensions(gen, m * n);
	hiprandGenerateLongLong(gen, dev_matrix, m * n);
	
	hipDeviceSynchronize();
	hipMemcpy(matrix, dev_matrix, m * n * sizeof(long long), hipMemcpyDeviceToHost);

	hipFree(dev_matrix);
}